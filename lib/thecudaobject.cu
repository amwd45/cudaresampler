#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "thrust/device_vector.h"
#include <vector>
#include "cudares_ff_impl.h"

__global__ void FIR_simpleFilterKernel(int nsamples,float* c_samples,
    float* cf_samples,float *c_taps,int ntaps)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;

    if(id < nsamples)
    {
        for(int t_index = 0;(t_index<ntaps)&&(id>=t_index);t_index++)
        {
            cf_samples[id] = cf_samples[id]+c_taps[t_index] * c_samples[id-t_index];
        }
    }
};


//void cu_FIR_Filter(int nsamples, int ntaps, float samples_h[], float f_samples_h[], float taps_h[])
void cu_FIR_Filter(std::vector<float> samples,int ntaps,std::vector<float> &f_samples, std::vector<float> taps)
{
    int deviceId,numberOfSMs;
    int threads_per_block,number_of_blocks;
    thrust::device_vector<float> c_samples(samples.begin(),samples.end());
    thrust::device_vector<float> cf_samples(f_samples.begin(),f_samples.end());
    thrust::device_vector<float> c_taps(taps.begin(),taps.end());
    float* cd_samples = thrust::raw_pointer_cast(&c_samples[0]);
    float* cd_taps = thrust::raw_pointer_cast(&c_taps[0]);
    float* cdf_samples = thrust::raw_pointer_cast(&cf_samples[0]);
    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs,hipDeviceAttributeMultiprocessorCount,deviceId);
    threads_per_block = 256;
    number_of_blocks = numberOfSMs*32;

    FIR_simpleFilterKernel<<<number_of_blocks,threads_per_block>>>(c_samples.size(),cd_samples,cdf_samples,cd_taps,ntaps);
    hipDeviceSynchronize();
    f_samples.clear();
    thrust::copy(cf_samples.begin(),cf_samples.end(),f_samples.begin());
    
};